
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define SIZE 10
/*
	to compile:
		submit.nvcc cuda-v-add.cu -o cuda-v-add
*/
//kernel function, will be loaded into the GPU (device)
__global__ void add_gpu(int* a, int* b, int* c, int size){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if( i < size ){				//if thread id exceed the array size....
		c[i] = a[i] + b[i];
	}
}
void add_cpu(int* a, int* b, int* c, int size){
	int i;
	for(i=0;i<size;i++){
		c[i] = a[i] + b[i];
	}
}
void fill(int* arr, int size){
	int i;
	for(i=0;i<size;i++){
		arr[i] = rand() % size;
	}
}
void display(int* arr, int size){
	int i;
	for(i=0;i<size;i++){
		printf("%d\t", arr[i]);
	}
	printf("\n");
}
int main(){
	//-------------------	host data allocation
	int *a, *b, *c;
	a = (int *) malloc(sizeof(int) * SIZE);
	b = (int *) malloc(sizeof(int) * SIZE);
	c = (int *) malloc(sizeof(int) * SIZE);
	fill(a, SIZE);
	fill(b, SIZE);
	//-------------------	operation
	add_cpu(a, b, c, SIZE);
	//------------------- 	display
	display(a, SIZE);
	display(b, SIZE);
	display(c, SIZE);
	//-------------------	device setup
	int *d_a, *d_b, *d_c;
	//allocate memory in device
	hipMalloc((void **)&d_a, sizeof(int) * SIZE);
	hipMalloc((void **)&d_b, sizeof(int) * SIZE);
	hipMalloc((void **)&d_c, sizeof(int) * SIZE);
	//copy input data from host to device
	hipMemcpy(d_a, a, sizeof(int) * SIZE, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, sizeof(int) * SIZE, hipMemcpyHostToDevice);
	//setup grid dim, and block dim
	int threads_per_block = (SIZE > 512) ? 512 : SIZE;
	int blocks_per_grid = ceil(SIZE / (double)threads_per_block);
	printf("GridDim: %d\tBlockDim: %d\n", blocks_per_grid, threads_per_block);
	//call the kernel function
	add_gpu<<<blocks_per_grid, threads_per_block>>>(d_a, d_b, d_c, SIZE);
	//copy output data from device to host
	hipMemcpy(c, d_c, sizeof(int) * SIZE, hipMemcpyDeviceToHost);

	display(c, SIZE);
	//-------------------	deallocation
	free(a);
	free(b);
	free(c);
	return 0;
}
